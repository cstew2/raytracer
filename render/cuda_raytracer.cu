#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdbool.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "render/cuda_raytracer.cuh"

__device__ int cuda_trace(const ray r, const scene *objects, bool shadow, hit_info *hi)
{
	float t_near = 1.0/0.0;
	float t;
	
	for(int i=0; i < objects->sphere_count; i++) {
		if(sphere_intersect(r, objects->spheres[i], &t)) {
			if(t_near > t) {
				t_near = t;
				sphere_hit(r, t, objects->spheres[i], hi);
			}
		}
	}

	for(int i=0; i < objects->plane_count; i++) {
		if(plane_intersect(r, objects->planes[i], &t)) {
			if(t_near > t) {
				t_near = t;
				plane_hit(r, t, objects->planes[i], hi);
			}
		}	
	}
	
	return t_near != 1.0/0.0;
}

__device__ vec4 cuda_cast_ray(const ray r, const scene *objects, const camera *cam)
{
	hit_info hi;
	cuda_trace(r, objects, false, &hi);

	vec4 c = vec4_new(0.0, 0.0, 0.0, 0.0);
	
	if(hi.hit_m.ambient > 0) {
		c = vec4_add(c, ambient(hi.hit_c, hi.hit_m.ambient));
	}

	if(hi.hit_m.diffuse  > 0 ||
	   hi.hit_m.specular > 0) {
		for(int l=0; l < objects->light_count; l++) {
			light l_i = objects->lights[l];
			vec3 l = vec3_scale(l_i.direction, -1.0);
			if(hi.hit_m.diffuse > 0) {
				vec4 d_colour = diffuse(hi.hit_n, hi.hit_c, hi.hit_m.diffuse,
							l, l_i.intensity, l_i.c);
				c = vec4_add(c, d_colour);
			}
		
			if(hi.hit_m.specular > 0) {
			
				vec4 s_colour = specular_reflection(hi.hit_n, hi.hit_c, hi.hit_m.diffuse,
								    l, l_i.intensity, l_i.c,
								    cam->position, 1.0);
				//c = vec4_add(c, s_colour);
			}
		}
	}
	
	return c;
}

__global__ void cuda_render_kernel(const scene *objects, camera *cam, canvas *can)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	ray r = generate_ray(cam, x, y);
	vec4 c = cuda_cast_ray(r, objects, cam);
	canvas_set_pixel(can, x, y, c);
}


#ifdef __NVCC__
extern "C" {
#endif

int cuda_render(raytracer rt)
{
	scene *objects_d = NULL;
	sphere *spheres_d = NULL;
	sphere *planes_d = NULL;
	sphere *triangles_d = NULL;
	sphere *lights_d = NULL;

	hipMalloc(&objects_d, sizeof(scene));
	hipMalloc(&spheres_d, sizeof(sphere)*rt.objects->sphere_count);
	hipMalloc(&planes_d, sizeof(plane)*rt.objects->plane_count);
	hipMalloc(&triangles_d, sizeof(triangle)*rt.objects->triangle_count);
	hipMalloc(&lights_d, sizeof(light)*rt.objects->light_count);
	
	hipMemcpy(objects_d, rt.objects,
		   sizeof(scene), hipMemcpyHostToDevice);
	hipMemcpy(spheres_d, rt.objects->spheres,
		   sizeof(sphere)*rt.objects->sphere_count, hipMemcpyHostToDevice);
	hipMemcpy(planes_d, rt.objects->planes,
		   sizeof(plane)*rt.objects->plane_count, hipMemcpyHostToDevice);
	hipMemcpy(triangles_d, rt.objects->triangles,
		   sizeof(triangle)*rt.objects->triangle_count, hipMemcpyHostToDevice);
	hipMemcpy(lights_d, rt.objects->lights,
		   sizeof(light)*rt.objects->light_count, hipMemcpyHostToDevice);	
	
	hipMemcpy(&(objects_d->spheres), &spheres_d, sizeof(sphere *), hipMemcpyHostToDevice);
	hipMemcpy(&(objects_d->planes), &planes_d, sizeof(plane *), hipMemcpyHostToDevice);
	hipMemcpy(&(objects_d->triangles), &triangles_d, sizeof(triangle *), hipMemcpyHostToDevice);
	hipMemcpy(&(objects_d->lights), &lights_d, sizeof(light *), hipMemcpyHostToDevice);
	
	camera *cam_d = NULL;
	hipMalloc(&cam_d, sizeof(camera));
	hipMemcpy(cam_d, &rt.camera, sizeof(camera), hipMemcpyHostToDevice);

	canvas *can_d = NULL;
	vec4 *screen_d = NULL;

	hipMalloc(&can_d, sizeof(canvas));
	hipMalloc(&screen_d, sizeof(vec4)*rt.canvas.width*rt.canvas.height);
	
	hipMemcpy(can_d, &rt.canvas, sizeof(canvas), hipMemcpyHostToDevice);

	hipMemcpy(screen_d, rt.canvas.screen,
		   sizeof(vec4)*rt.canvas.width*rt.canvas.height, hipMemcpyHostToDevice);
	hipMemcpy(&(can_d->screen), &screen_d, sizeof(vec4 *), hipMemcpyHostToDevice);
	
	dim3 threads(8, 8);
	dim3 blocks(rt.canvas.width/threads.x,  
		    rt.canvas.height/threads.y);
	
	cuda_render_kernel<<<blocks, threads>>>(objects_d, cam_d, can_d);

	hipMemcpy(rt.canvas.screen, screen_d,
		   sizeof(vec4)*rt.canvas.width*rt.canvas.height, hipMemcpyDeviceToHost);

	write_ppm_file("frame.pnm", rt.canvas);
	
	hipFree(screen_d);
	hipFree(can_d);
	hipFree(cam_d);
	hipFree(lights_d);
	hipFree(triangles_d);
	hipFree(planes_d);
	hipFree(spheres_d);
	hipFree(objects_d);
	
	return 0;
}
	
#ifdef __NVCC__
}
#endif	
